#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <memory>
#include <cmath>
#include <map>
#include <functional>
#include <py_helper.hpp>
#include "kernels.hpp"

// __global__ void PDE_diffusion_equation(real* out, const real* in, const real dx, const real dy, const real dt, const real D, int nx, int ny, const int nz = 1){
//     get_thread_index_3d(x, y, z);
//     if (!(boundary_check(x, nx) && boundary_check(y, ny) && boundary_check(z, nz))) return;
//     for(int i = 0; i < nz; i ++){
//         int page_offset = i *(nx * ny);
//         out[page_offset + y * nx + x] = in[y * nx + x] + dt * D * laplace(in, x, y, nx, dx, dy);
//         in = out + page_offset; 
//         __syncthreads();
//     }
// }


// void solve_diffusion_with_pde(real dt, real simulate_time, real Lx, real Ly, real dx, real dy,real sigmax, real sigmay, real D, std::function<bool(np::ndarray)> callback = nullptr) {
//     const int nx = std::ceil(Lx / dx);
//     const int ny = std::ceil(Ly / dy);
//     const int nz = 1;
//     const int iter_count = static_cast<int>(simulate_time / dt) / nz + 1;
//     const dim3 blockSize(16, 16);
//     const dim3 gridSize(aligin_count(nx, blockSize.x), aligin_count(ny, blockSize.y), nz);

//     cuda_vector_device<real> mem_device_in; mem_device_in.reserve(nx * ny * nz);
//     cuda_vector_device<real> mem_device_out; mem_device_out.reserve(nx * ny * nz);
//     cuda_vector_host<real> result(nx * ny * nz);

//     real *f = mem_device_out.data(), *f0 = mem_device_in.data();
//     gauss_kernel << <gridSize, blockSize >> > (f0, dx, dy, sigmax, sigmay, 1.0 /*gauss_normalize(sigmax * nx, sigmax * nx)*/, nx, ny, nz);
//     hipDeviceSynchronize(); check_cuda_error();
//     hipMemcpy(result.data(), f0, result.size() * sizeof(real), hipMemcpyDeviceToHost);

//     if(!callback(create_ndarray_from_vector(result, { nx, ny }))) return;
//     for (int k = 0; k < iter_count; ++k) {
//         if (result.end() != std::find_if(result.begin(), result.end(), [](real n) {return std::isnan(n); })) {
//             return;
//         }
//         PDE_diffusion_equation << <gridSize, blockSize >> > (f, f0, dx, dy, dt, D, nx, ny, nz);
//         hipDeviceSynchronize();

//         hipMemcpy(result.data(), f, result.size() * sizeof(real), hipMemcpyDeviceToHost);
//         if (!callback(create_ndarray_from_vector(result, { nx, ny }))) return;
//         //== set f as f0 for next iter
//         std::swap(f, f0);
//     }
// }

// //////////////////////////////
// struct sigmoid_solver{
//     real scalar{1}, bais{0};
//     real mapping_accuracy_to_span(const real min, const real max, const real accuracy = 1e-2) {
//         //== mapping x from [-3, 3] to [0.05, 0.95]
//         const std::map<real, real> mapping_accuracy_to_span{ {1e-2f, 3.0f} };
//         assert(std::abs((max - min) / 2) > accuracy * 9);
//         return mapping_accuracy_to_span.at(accuracy);
//     }
//     __both_side__ void mapping_to(const real min, const real max, const real span = 3.0f) {
//         bais = (min + max) / 2;
//         scalar = span / ((max - min) / 2);
//     }
//     __both_side__ real solve(real x){
//         //== y = \frac{1}{1 + e^(scalar(-x) + bais)}
//         return 1.0 / (1 + std::exp(-scalar * (x + bais)));
//     }
// };
// __global__ void initialize_kernel(real* f, int nx, int ny){
//     get_thread_index_2d(idx, idy);
//     if(!(boundary_check(idx, nx) && boundary_check(idy, ny))) return; 
//     auto cal_delta =[](int n)->real{return 1.0/(n - 1);};
//     real dx = cal_delta(nx); 
//     real dy = cal_delta(ny); 

//     real x = idx * dx;
//     real y = idy * dy;

//     const real min_boundary = 0.4;
//     const real max_boundary = 0.6;
//     const real smooth_ratio = 0.75;
//     sigmoid_solver left_boundary,right_boundary;
//     left_boundary.mapping_to(min_boundary * smooth_ratio, min_boundary);
//     right_boundary.mapping_to(max_boundary + (1.0 - max_boundary) * smooth_ratio, max_boundary);
//     auto cal_f = [&](real n)->real {
//         if (min_boundary < n && n < max_boundary) return 1.0;
//         else if (min_boundary >= n) return left_boundary.solve(n);
//         else return right_boundary.solve(n);
//     };
//     f[idy * nx + idx] = cal_f(x) * cal_f(y);
// }
// __global__ void green_function_kernel(real* g, const int nx, const int ny, real t, const real D = 1.0) {
//     get_thread_index_2d(idx, idy);
//     if(!(boundary_check(idx, nx) && boundary_check(idy, ny))) return; 
//     auto cal_delta =[](int n)->real{return 1.0/ n;};
//     real x = idx * cal_delta(nx); 
//     real y = idy * cal_delta(ny); 

//     real sigma = std::sqrt(2.0 * D * t) * nx;
//     g[idy * nx + idx] = gauss(idx, idy, 0.5 * nx, 0.5 * ny, sigma, gauss_normalize(sigma));
// }
// void solve_diffusion_with_green_function(real dt, real simulate_time, real Lx, real Ly, real dx, real dy, real sigmax, real sigmay, real D, std::function<bool(np::ndarray)> callback) {
//     const int nx = std::ceil(Lx / dx);
//     const int ny = std::ceil(Ly / dy);
//     const int nz = 1;
//     const int iter_count = static_cast<int>(simulate_time / dt) / nz + 1;
//     const dim3 blockSize(16, 16);
//     const dim3 gridSize(aligin_count(nx, blockSize.x), aligin_count(ny, blockSize.y), nz);

//     auto f0 = make_inplace_fft_vec<cuda_allocator<real, cuda_memory_type::device>>(nx, ny);
//     auto g = make_inplace_fft_vec<cuda_allocator<real, cuda_memory_type::device>>(nx, ny);

//     gauss_kernel << <gridSize, blockSize >> > (f0.data(), dx, dy, sigmax, sigmay, 1.0 /*gauss_normalize(sigmax * nx, sigmax * nx)*/, nx, ny, nz);
//     hipDeviceSynchronize(); check_cuda_error();

//     cuda_vector_host<real> result; result << f0;
//     if (!callback(create_ndarray_from_vector(result, { nx, ny }))) return;
//     hipDeviceSynchronize(); check_cuda_error();
//     for (int k = 1; k < iter_count; ++k) {
//         green_function_kernel << <gridSize, blockSize >> > (g.data(), nx, ny, dt);
//         hipDeviceSynchronize(); check_cuda_error();
//         fft_convolve(f0, f0, g, nx, ny);
//         hipDeviceSynchronize(); check_cuda_error();
//         result << f0;
//         if (!callback(create_ndarray_from_vector(result, { nx, ny }))) return;
//     }
// }

// int main() {
//     py_loader::init();
//     struct diffusion_input_params {
//         const real D = 1;     // 扩散系数
//         const real dx = 0.001;   // 空间步长
//         const real dy = 0.001;   // 空间步长
//         const real Lx = 1.0;    // 区域长度
//         const real Ly = 1.0;    // 区域宽度
//         const real dt = 0.001;  // 时间步长
//         const real T = 1.0;     // 仿真时间
//         const real sigmax = 0.1;// 高斯分布标准差
//         const real sigmay = 0.1;
//     }param;
//     bool use_pde = true;
//     if(use_pde){
//         solve_diffusion_with_pde(
//             param.dt, param.T, 
//             param.Lx, param.Ly, 
//             param.dx, param.dy, 
//             param.sigmax, param.sigmay,
//             param.D, py_plot::create_callback_simulation_fram_done()
//         );
//     }
//     else{
//         solve_diffusion_with_green_function(
//             param.dt, param.T,
//             param.Lx, param.Ly,
//             param.dx, param.dy,
//             param.sigmax, param.sigmay,
//             param.D, py_plot::create_callback_simulation_fram_done()
//         );
//     }
//     return 0;
// }

int main()
{
    
}