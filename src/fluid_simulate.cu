#include "hip/hip_runtime.h"
#include "kernels.hpp"
#include <py_helper.hpp>
#include <cuda_vector.hpp>
//__global__ void addForces_k(float2 *v, int nx, int ny, float fx, float fy, float effect_len) {
//    get_thread_index_2d(ix, iy);
//    if(!(boundary_check(ix, nx, r) && boundary_check(iy, ny, r))) return;
//    real tx = ix - r;
//    real ty = iy - r;
//    float s = 1.f / (1.f + tx * tx * tx * tx + ty * ty * ty * ty);
//    float2& vterm = v[nx * iy + ix];
//    vterm.x += s * fx;
//    vterm.y += s * fy;
//}
int main() {
    py_engine::init();
    int nx = 100, ny = 100;
    cuda::pageable_vector<real> vec(nx * ny);
    auto callback = py_plot::create_callback_simulation_fram_done(py::object(overload_click));
    while (callback(create_ndarray_from_vector(vec, { nx, ny })));
}